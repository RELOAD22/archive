#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <algorithm>
#include <hip/device_functions.h>
#include <pthread.h>
#include <sys/time.h>
#include <thread>
#include <iostream>
#include <unistd.h>
#include "common.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <vector>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <algorithm>
#include <hip/device_functions.h>
#include <pthread.h>
#include <sys/time.h>
#include <thread>
#include <iostream>
using namespace std;

#define ITER 1

#define NELEM 32
#define WIDTH 32
#define ITEMS_PER_THREAD 8
#define M 1024
#define N 1024
#define K 1024
int nElem = 1 << 28;
float *d_a, *d_b, *d_c;
float *d_A, *d_B, *d_C;
#define gridsize (max(M, N) + WIDTH - 1) / WIDTH
dim3 grid(gridsize, gridsize);
dim3 block(WIDTH, WIDTH / ITEMS_PER_THREAD);

#define CONTEXT_POOL_SIZE 6
hipCtx_t contextPool[CONTEXT_POOL_SIZE];
int smCounts[CONTEXT_POOL_SIZE];

class TimeInterval
{
public:
    TimeInterval() : start_(std::chrono::steady_clock::now()) {}

    double Elapsed()
    {
        auto now = std::chrono::steady_clock::now();
        return std::chrono::duration_cast<Duration>(now - start_).count();
    }

private:
    using Duration = std::chrono::duration<double>;
    std::chrono::steady_clock::time_point start_;
};

void print_current_affinity()
{
    CUexecAffinityParam affinity_new;
    cuCtxGetExecAffinity(&affinity_new, CU_EXEC_AFFINITY_TYPE_SM_COUNT);
    int numSms = affinity_new.param.smCount.val;
    printf("current affinity.param.smCount.val:%d\n", numSms);
}

// A:M*K B:K*N C:M*N
void CpuMatmul(float *a, float *b, float *c, int m, int n, int k)
{
    float sum = 0;
    for (int i = 0; i < m; ++i)
    {
        for (int j = 0; j < n; ++j)
        {
            sum = 0;
            for (int t = 0; t < k; ++t)
            {
                sum += a[i * k + t] * b[t * n + j];
            }
            c[i * n + j] = sum;
        }
    }
}

void InitData(float *a, float *b, int m, int n, int k)
{
    for (int i = 0; i < m; ++i)
    {
        for (int j = 0; j < k; ++j)
        {
            *(a + i * k + j) = 2;
        }
    }
    for (int i = 0; i < k; ++i)
    {
        for (int j = 0; j < n; ++j)
        {
            *(b + i * n + j) = 3;
        }
    }
}
__global__ void matmul(float *a, float *b, float *c, int m, int n, int k)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y * ITEMS_PER_THREAD + threadIdx.y;

    // int threadid = threadIdx.x + threadIdx.y * blockDim.x;
    int tid = y * n + x;
    if (x >= n || y >= m)
        return;
    __shared__ float sha[WIDTH][WIDTH];
    __shared__ float shb[WIDTH][WIDTH];

    // as[tid] = a[]
    float sum[ITEMS_PER_THREAD] = {0};
    int nIter = (k + WIDTH - 1) / WIDTH;

    for (int i = 0; i < nIter; i++)
    {
        for (int index = 0; index < ITEMS_PER_THREAD; ++index)
        {
            int offset = index * (WIDTH / ITEMS_PER_THREAD);
            sha[threadIdx.y + offset][threadIdx.x] = a[(y + offset) * k + i * WIDTH + threadIdx.x];
            shb[threadIdx.y + offset][threadIdx.x] = b[(i * WIDTH + threadIdx.y + offset) * n + x];
        }
        /*
        sha[threadIdx.y][threadIdx.x] = a[y * k + i * WIDTH + threadIdx.x];
        shb[threadIdx.y][threadIdx.x] = b[(i * WIDTH + threadIdx.y) * n + x];
        sha[threadIdx.y + 8][threadIdx.x] = a[(y + 8) * k + i * WIDTH + threadIdx.x];
        shb[threadIdx.y + 8][threadIdx.x] = b[(i * WIDTH + threadIdx.y + 8) * n + x];
        sha[threadIdx.y + 16][threadIdx.x] = a[(y + 16) * k + i * WIDTH + threadIdx.x];
        shb[threadIdx.y + 16][threadIdx.x] = b[(i * WIDTH + threadIdx.y + 16) * n + x];
        sha[threadIdx.y + 24][threadIdx.x] = a[(y + 24) * k + i * WIDTH + threadIdx.x];
        shb[threadIdx.y + 24][threadIdx.x] = b[(i * WIDTH + threadIdx.y + 24) * n + x];
        */
        __syncthreads();
        for (int index = 0; index < WIDTH; ++index)
        {
            for (int i = 0; i < ITEMS_PER_THREAD; ++i)
            {
                int offset = i * (WIDTH / ITEMS_PER_THREAD);
                sum[i] += sha[threadIdx.y + offset][index] * shb[index][threadIdx.x];
            }
            /*
            sum[0] += sha[threadIdx.y][index] * shb[index][threadIdx.x];
            sum[1] += sha[threadIdx.y + 8][index] * shb[index][threadIdx.x];
            sum[2] += sha[threadIdx.y + 16][index] * shb[index][threadIdx.x];
            sum[3] += sha[threadIdx.y + 24][index] * shb[index][threadIdx.x];
            */
            // printf("tid: %d read: a[%d],b[%d]\n", tid, y * size + index, index * size + x);
        }
        __syncthreads();
    }

    for (int index = 0; index < ITEMS_PER_THREAD; ++index)
    {
        int offset = index * (WIDTH / ITEMS_PER_THREAD);
        c[(y + offset) * n + x] = sum[index];
    }
    /*
    c[tid] = sum[0];
    c[(y + 8) * n + x] = sum[1];
    c[(y + 16) * n + x] = sum[2];
    c[(y + 24) * n + x] = sum[3];
    */
    // printf("c[%d]:%f size:%d\n", tid, sum, size);
}
hipDeviceProp_t prop;
// hipCtx_t contextPool;

void initialData(float *ip, int size)
{
    // generate different seed for random number
    time_t t;
    srand((unsigned)time(&t));

    for (int i = 0; i < size; i++)
    {
        ip[i] = (float)(rand() & 0xFF) / 10.0f;
    }

    return;
}

void sumArraysOnHost(float *A, float *B, float *C, const int Ns)
{
    for (int idx = 0; idx < Ns; idx++)
    {
        C[idx] = A[idx] + B[idx];
    }
}
__global__ void sumArraysOnGPU(float *A, float *B, float *C, const int Ns)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    while (i < Ns)
    {
        C[i] = A[i] + 1;
        i += gridDim.x * blockDim.x;
    }
}

void *thread1(void *t)
{
    printf("thread1:\n");
    hipCtxSetCurrent(contextPool[0]);

    auto mata_size = M * K * sizeof(float);
    auto matb_size = K * N * sizeof(float);
    auto matc_size = M * N * sizeof(float);

    auto elems_size = NELEM * NELEM;

    float *a = (float *)malloc(mata_size);
    float *b = (float *)malloc(matb_size);
    float *c = (float *)malloc(matc_size);
    float *cpu_c = (float *)malloc(matc_size);

    InitData(a, b, M, N, K);
    float *d_a, *d_b, *d_c;
    hipMalloc(&d_a, mata_size);
    hipMalloc(&d_b, matb_size);
    hipMalloc(&d_c, matc_size);
    hipMemset(d_c, 0, matc_size);

    hipMemcpy(d_a, a, mata_size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, matb_size, hipMemcpyHostToDevice);
    dim3 grid(gridsize, gridsize);

    dim3 block(WIDTH, WIDTH / ITEMS_PER_THREAD);

    print_current_affinity();
    struct timeval start_time, end_time;
    gettimeofday(&start_time, NULL);
    for (int i = 0; i < ITER; ++i)
        matmul<<<grid, block> > >(d_a, d_b, d_c, M, N, K);
    hipDeviceSynchronize();
    gettimeofday(&end_time, NULL);
    double timespend = (end_time.tv_sec * 1000000 + end_time.tv_usec) - (start_time.tv_sec * 1000000 + start_time.tv_usec);
    printf("time:%f\n", timespend);

    hipMemcpy(c, d_c, matc_size, hipMemcpyDeviceToHost);

    printf("%f ", *(c));
    printf("%f ", *(c + M * N / 2 + N / 2));
    printf("%f ", *(c + M * N - 1));
    printf("\n");
}

void *mythread(int i)
{
    hipCtxSetCurrent(contextPool[i]);
    print_current_affinity();
    TimeInterval T;
    for (int i = 0; i < ITER; ++i)
        sumArraysOnGPU<<<grid, block> > >(d_A, d_B, d_C, nElem);
    hipDeviceSynchronize();
    auto stop = T.Elapsed();
    std::cout << "in " << stop << " seconds" << std::endl;
}

int main()
{

    // set up data size of vectors
    int nElem = 1 << 28;
    printf("Vector size %d\n", nElem);

    // malloc host memory
    size_t nBytes = nElem * sizeof(float);

    float *h_A, *h_B, *hostRef, *gpuRef;
    h_A = (float *)malloc(nBytes);
    h_B = (float *)malloc(nBytes);
    hostRef = (float *)malloc(nBytes);
    gpuRef = (float *)malloc(nBytes);

    printf("init finish\n");

    // CpuMatmul(a, b, cpu_c, M, N, K);
    printf("cpu compute finish\n");

    // init device data
    int device = 0;

    CHECK(hipMalloc((float **)&d_A, nBytes));
    CHECK(hipMalloc((float **)&d_B, nBytes));
    CHECK(hipMalloc((float **)&d_C, nBytes));

    printf("begin kernel:\n");
    printf("grid:%d %d \n", gridsize, gridsize);

    CUexecAffinityParam affinity[2];
    affinity[0].type = CU_EXEC_AFFINITY_TYPE_SM_COUNT;
    affinity[0].param.smCount.val = (unsigned int)2;
    affinity[1].type = CU_EXEC_AFFINITY_TYPE_MAX;
    affinity[1].param.smCount.val = (unsigned int)34;
    hipSetDevice(device);
    printf("cuCtxCreate_v3\n");
    cuCtxCreate_v3(&contextPool[0], affinity, 1, 0, 0);

    print_current_affinity();

    hipGetDeviceProperties(&prop, device);
    printf("current prop.multiProcessorCount:%d\n", prop.multiProcessorCount);

    // printf("hipCtxSetCurrent finish\n");
    // pthread_t pt;
    // pthread_create(&pt, NULL, thread1, NULL);
    // pthread_join(pt, NULL);

    /*
    smCounts[0] = (prop. multiProcessorCount - 5) / 5 * 1;
    smCounts[1] = (prop. multiProcessorCount - 5) / 5 * 2;
    smCounts[2] = (prop. multiProcessorCount - 5) / 5 * 3;
    smCounts[3] = (prop. multiProcessorCount - 5) / 5 * 4;
    smCounts[4] = (prop. multiProcessorCount);*/

    for (int i = 0; i < CONTEXT_POOL_SIZE; i++)
    {
        smCounts[i] = (prop.multiProcessorCount - 5) / CONTEXT_POOL_SIZE * (i + 1);
    }
    smCounts[CONTEXT_POOL_SIZE - 1] = (prop.multiProcessorCount);
    /*
    for (int i = 0; i < CONTEXT_POOL_SIZE; i++){
        smCounts[i] = (prop. multiProcessorCount);
    }*/

    for (int i = 0; i < CONTEXT_POOL_SIZE; i++)
    {
        CUexecAffinityParam affinity[2];
        affinity[0].type = CU_EXEC_AFFINITY_TYPE_SM_COUNT;
        affinity[0].param.smCount.val = smCounts[i];
        cuCtxCreate_v3(&contextPool[i], affinity, 1, 0, 0);
    }

    usleep(500000);
    printf("begin kernel:\n");
    TimeInterval T;
    pthread_t pts[CONTEXT_POOL_SIZE];
    for (int i = 0; i < CONTEXT_POOL_SIZE; i++)
    {
        /*
        std::thread([i]() {
            printf("thread%d:\n", i);
            CUexecAffinityParam affinity;
            hipCtxSetCurrent(contextPool[i]);
            cuCtxGetExecAffinity(&affinity, CU_EXEC_AFFINITY_TYPE_SM_COUNT);
            numSms = affinity.param.smCount.val;
            printf("numSms:%d  numBlocksPerSm:%d\n", numSms, numBlocksPerSm);
            void *kernelArgs[] = { };
            hipLaunchCooperativeKernel((void*)matmul, dimGrid, dimBlock, kernelArgs);
        });*/
        std::thread t(mythread, i);
        // pthread_create(&pts[i], NULL, mythread, (void *)i);
        t.join();
    }

    print_current_affinity();

    for (int i = 0; i < ITER; ++i)
        sumArraysOnGPU<<<grid, block> > >(d_A, d_B, d_C, nElem);
    hipDeviceSynchronize();
    auto stop = T.Elapsed();
    std::cout << "in" << stop << "seconds" << std::endl;

    /*
    for(int i = 0; i < 64; ++i){
        for(int j = 0; j < 64; ++j)
            printf("%f ", *(c + i * N + j));
        printf("\n");
    }*/
    /*
    for (int i = 0; i < M; ++i)
    {
        for (int j = 0; j < N; ++j)
        {
            if (cpu_c[i * N + j] != c[i * N + j])
            {
                printf("ERROR AT: c[%d][%d]: %f(cpu) %f(gpu)", i, j, cpu_c[i * N + j], c[i * N + j]);
                return 0;
            }
        }
    }*/
    printf("\n");
}