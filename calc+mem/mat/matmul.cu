#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <algorithm>
#include <hip/device_functions.h>
using namespace std;
#define NELEM 32
#define WIDTH 32
#define ITEMS_PER_THREAD 8
#define M 1024
#define N 1024
#define K 1024

// A:M*K B:K*N C:M*N
void CpuMatmul(float *a, float *b, float *c, int m, int n, int k)
{
    float sum = 0;
    for (int i = 0; i < m; ++i)
    {
        for (int j = 0; j < n; ++j)
        {
            sum = 0;
            for (int t = 0; t < k; ++t)
            {
                sum += a[i * k + t] * b[t * n + j];
            }
            c[i * n + j] = sum;
        }
    }
}

void InitData(float *a, float *b, int m, int n, int k)
{
    for (int i = 0; i < m; ++i)
    {
        for (int j = 0; j < k; ++j)
        {
            *(a + i * k + j) = 2;
        }
    }
    for (int i = 0; i < k; ++i)
    {
        for (int j = 0; j < n; ++j)
        {
            *(b + i * n + j) = 3;
        }
    }
}
__global__ void matmul(float *a, float *b, float *c, int m, int n, int k)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y * ITEMS_PER_THREAD + threadIdx.y;

    // int threadid = threadIdx.x + threadIdx.y * blockDim.x;
    int tid = y * n + x;
    if (x >= n || y >= m)
        return;
    __shared__ float sha[WIDTH][WIDTH];
    __shared__ float shb[WIDTH][WIDTH];

    // as[tid] = a[]
    float sum[ITEMS_PER_THREAD] = {0};
    int nIter = (k + WIDTH - 1) / WIDTH;

    for (int i = 0; i < nIter; i++)
    {
        for (int index = 0; index < ITEMS_PER_THREAD; ++index)
        {
            int offset = index * (WIDTH / ITEMS_PER_THREAD);
            sha[threadIdx.y + offset][threadIdx.x] = a[(y + offset) * k + i * WIDTH + threadIdx.x];
            shb[threadIdx.y + offset][threadIdx.x] = b[(i * WIDTH + threadIdx.y + offset) * n + x];
        }
        /*
        sha[threadIdx.y][threadIdx.x] = a[y * k + i * WIDTH + threadIdx.x];
        shb[threadIdx.y][threadIdx.x] = b[(i * WIDTH + threadIdx.y) * n + x];
        sha[threadIdx.y + 8][threadIdx.x] = a[(y + 8) * k + i * WIDTH + threadIdx.x];
        shb[threadIdx.y + 8][threadIdx.x] = b[(i * WIDTH + threadIdx.y + 8) * n + x];
        sha[threadIdx.y + 16][threadIdx.x] = a[(y + 16) * k + i * WIDTH + threadIdx.x];
        shb[threadIdx.y + 16][threadIdx.x] = b[(i * WIDTH + threadIdx.y + 16) * n + x];
        sha[threadIdx.y + 24][threadIdx.x] = a[(y + 24) * k + i * WIDTH + threadIdx.x];
        shb[threadIdx.y + 24][threadIdx.x] = b[(i * WIDTH + threadIdx.y + 24) * n + x];
        */
        __syncthreads();
        for (int index = 0; index < WIDTH; ++index)
        {
            for (int i = 0; i < ITEMS_PER_THREAD; ++i)
            {
                int offset = i * (WIDTH / ITEMS_PER_THREAD);
                sum[i] += sha[threadIdx.y + offset][index] * shb[index][threadIdx.x];
            }
            /*
            sum[0] += sha[threadIdx.y][index] * shb[index][threadIdx.x];
            sum[1] += sha[threadIdx.y + 8][index] * shb[index][threadIdx.x];
            sum[2] += sha[threadIdx.y + 16][index] * shb[index][threadIdx.x];
            sum[3] += sha[threadIdx.y + 24][index] * shb[index][threadIdx.x];
            */
            // printf("tid: %d read: a[%d],b[%d]\n", tid, y * size + index, index * size + x);
        }
        __syncthreads();
    }

    for (int index = 0; index < ITEMS_PER_THREAD; ++index)
    {
        int offset = index * (WIDTH / ITEMS_PER_THREAD);
        c[(y + offset) * n + x] = sum[index];
    }
    /*
    c[tid] = sum[0];
    c[(y + 8) * n + x] = sum[1];
    c[(y + 16) * n + x] = sum[2];
    c[(y + 24) * n + x] = sum[3];
    */
    // printf("c[%d]:%f size:%d\n", tid, sum, size);
}

int main()
{

    auto mata_size = M * K * sizeof(float);
    auto matb_size = K * N * sizeof(float);
    auto matc_size = M * N * sizeof(float);

    auto elems_size = NELEM * NELEM;

    float *a = (float *)malloc(mata_size);
    float *b = (float *)malloc(matb_size);
    float *c = (float *)malloc(matc_size);
    float *cpu_c = (float *)malloc(matc_size);

    InitData(a, b, M, N, K);
    printf("init finish\n");

    // CpuMatmul(a, b, cpu_c, M, N, K);
    printf("cpu compute finish\n");

    float *d_a, *d_b, *d_c;
    hipMalloc(&d_a, mata_size);
    hipMalloc(&d_b, matb_size);
    hipMalloc(&d_c, matc_size);
    hipMemset(d_c, 0, matc_size);

    hipMemcpy(d_a, a, mata_size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, matb_size, hipMemcpyHostToDevice);
    printf("begin kernel:\n");
    int gridsize = (max(M, N) + WIDTH - 1) / WIDTH;
    printf("grid:%d %d \n", gridsize, gridsize);
    dim3 grid(gridsize, gridsize);

    dim3 block(WIDTH, WIDTH / ITEMS_PER_THREAD);
    matmul<<<grid, block> > >(d_a, d_b, d_c, M, N, K);
    hipDeviceSynchronize();

    hipMemcpy(c, d_c, matc_size, hipMemcpyDeviceToHost);
    /*
    for(int i = 0; i < 64; ++i){
        for(int j = 0; j < 64; ++j)
            printf("%f ", *(c + i * N + j));
        printf("\n");
    }*/
    /*
    for (int i = 0; i < M; ++i)
    {
        for (int j = 0; j < N; ++j)
        {
            if (cpu_c[i * N + j] != c[i * N + j])
            {
                printf("ERROR AT: c[%d][%d]: %f(cpu) %f(gpu)", i, j, cpu_c[i * N + j], c[i * N + j]);
                return 0;
            }
        }
    }*/
    printf("%f ", *(c));
    printf("%f ", *(c + M * N / 2 + N / 2));
    printf("%f ", *(c + M * N - 1));
    printf("\n");
}