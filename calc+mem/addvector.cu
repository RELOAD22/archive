#include "hip/hip_runtime.h"
#include "common.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <vector>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <algorithm>
#include <hip/device_functions.h>
#include <pthread.h>
#include <sys/time.h>
#include <thread>
#include <iostream>

class TimeInterval
{
public:
    TimeInterval() : start_(std::chrono::steady_clock::now()) {}

    double Elapsed()
    {
        auto now = std::chrono::steady_clock::now();
        return std::chrono::duration_cast<Duration>(now - start_).count();
    }

private:
    using Duration = std::chrono::duration<double>;
    std::chrono::steady_clock::time_point start_;
};

void checkResult(float *hostRef, float *gpuRef, const int N)
{
    double epsilon = 1.0E-8;
    bool match = 1;

    for (int i = 0; i < N; i++)
    {
        if (abs(hostRef[i] - gpuRef[i]) > epsilon)
        {
            match = 0;
            printf("Arrays do not match!\n");
            printf("host %5.2f gpu %5.2f at current %d\n", hostRef[i],
                   gpuRef[i], i);
            break;
        }
    }

    if (match)
        printf("Arrays match.\n\n");

    return;
}

void initialData(float *ip, int size)
{
    // generate different seed for random number
    time_t t;
    srand((unsigned)time(&t));

    for (int i = 0; i < size; i++)
    {
        ip[i] = (float)(rand() & 0xFF) / 10.0f;
    }

    return;
}

void sumArraysOnHost(float *A, float *B, float *C, const int N)
{
    for (int idx = 0; idx < N; idx++)
    {
        C[idx] = A[idx] + B[idx];
    }
}
__global__ void sumArraysOnGPU(float *A, float *B, float *C, const int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    while (i < N)
    {
        C[i] = A[i] + 1;
        i += gridDim.x * blockDim.x;
    }
}

void print_current_affinity()
{
    CUexecAffinityParam affinity_new;
    cuCtxGetExecAffinity(&affinity_new, CU_EXEC_AFFINITY_TYPE_SM_COUNT);
    int numSms = affinity_new.param.smCount.val;
    printf("current affinity.param.smCount.val:%d\n", numSms);
}

int main(int argc, char **argv)
{
    printf("%s Starting...\n", argv[0]);

    // set up device
    int dev = 0;

    CUexecAffinityParam affinity[1];
    hipCtx_t contextPool;
    hipDeviceProp_t prop;
    affinity[0].type = CU_EXEC_AFFINITY_TYPE_SM_COUNT;
    affinity[0].param.smCount.val = (unsigned int)32;

    hipSetDevice(dev);
    printf("cuCtxCreate_v3\n");
    cuCtxCreate_v3(&contextPool, affinity, 1, 0, 0);

    print_current_affinity();

    hipGetDeviceProperties(&prop, dev);
    printf("current prop.multiProcessorCount:%d\n", prop.multiProcessorCount);

    // set up data size of vectors
    int nElem = 1 << 28;
    printf("Vector size %d\n", nElem);

    // malloc host memory
    size_t nBytes = nElem * sizeof(float);

    float *h_A, *h_B, *hostRef, *gpuRef;
    h_A = (float *)malloc(nBytes);
    h_B = (float *)malloc(nBytes);
    hostRef = (float *)malloc(nBytes);
    gpuRef = (float *)malloc(nBytes);

    double iStart, iElaps;

    // initialize data at host side
    iStart = seconds();
    // initialData(h_A, nElem);
    // initialData(h_B, nElem);
    iElaps = seconds() - iStart;
    printf("initialData Time elapsed %f sec\n", iElaps);
    memset(hostRef, 0, nBytes);
    memset(gpuRef, 0, nBytes);

    // malloc device global memory
    float *d_A, *d_B, *d_C;
    CHECK(hipMalloc((float **)&d_A, nBytes));
    CHECK(hipMalloc((float **)&d_B, nBytes));
    CHECK(hipMalloc((float **)&d_C, nBytes));

    // transfer data from host to device
    // CHECK(hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice));
    // CHECK(hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice));
    // CHECK(hipMemcpy(d_C, gpuRef, nBytes, hipMemcpyHostToDevice));

    // std::vector<int> blocks = {1024, 512 , 256, 128, 64};
    std::vector<int> blocks = {1024};

    printf("begin kernel:\n");
    TimeInterval T;
    for (int iter = 0; iter < 1; ++iter)
        for (auto blocksize : blocks)
        {
            // invoke kernel at host side
            int iLen = blocksize;
            dim3 block(iLen);
            // dim3 grid  ((nElem + block.x - 1) / (block.x ));
            dim3 grid(32);

            // iStart = seconds();
            sumArraysOnGPU<<<grid, block> > >(d_A, d_B, d_C, nElem);
            CHECK(hipDeviceSynchronize());
            // iElaps = seconds() - iStart;
            // printf("sumArraysOnGPU <<<  %d, %d  >>>  Time elapsed %f sec\n", grid.x,
            // block.x, iElaps);
        }

    hipDeviceSynchronize();
    auto stop = T.Elapsed();
    std::cout << "in" << stop << "seconds" << std::endl;

    // check kernel error
    CHECK(hipGetLastError());

    // copy kernel result back to host side
    CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));

    // check device results
    // checkResult(hostRef, gpuRef, nElem);

    // free device global memory
    CHECK(hipFree(d_A));
    CHECK(hipFree(d_B));
    CHECK(hipFree(d_C));

    // free host memory
    free(h_A);
    free(h_B);
    free(hostRef);
    free(gpuRef);

    return (0);
}
